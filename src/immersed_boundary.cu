#include "hip/hip_runtime.h"
/*
 * MicroHH
 * Copyright (c) 2011-2020 Chiel van Heerwaarden
 * Copyright (c) 2011-2020 Thijs Heus
 * Copyright (c) 2014-2020 Bart van Stratum
 *
 * This file is part of MicroHH
 *
 * MicroHH is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.

 * MicroHH is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.

 * You should have received a copy of the GNU General Public License
 * along with MicroHH.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <iostream>

#include "immersed_boundary.h"
#include "boundary.h"
#include "fields.h"
#include "tools.h"
#include "fast_math.h"

namespace
{
    namespace fm = Fast_math;
    
    template<typename TF> __global__
    void set_forcing_points_u_g(
        TF* const __restrict__ tend_u,
        const TF* const __restrict__ tend_v,
        const TF* const __restrict__ tend_w,
        const TF* const __restrict__ fld_u,
        const TF* const __restrict__ fld_v,
        const TF* const __restrict__ fld_w,
        const TF* const __restrict__ boundary_value,
        const int* const __restrict__ gi, const int* const __restrict__ gj, const int* const __restrict__ gk,
        const TF* const __restrict__ rot,
        const int* const __restrict__ ipui, const int* const __restrict__ ipuj, const int* const __restrict__ ipuk, const TF* const __restrict__ c_idw_u,
        const int* const __restrict__ ipvi, const int* const __restrict__ ipvj, const int* const __restrict__ ipvk, const TF* const __restrict__ c_idw_v, 
        const int* const __restrict__ ipwi, const int* const __restrict__ ipwj, const int* const __restrict__ ipwk, const TF* const __restrict__ c_idw_w,
        const int* const __restrict__ ipsi, const int* const __restrict__ ipsj, const int* const __restrict__ ipsk, const TF* const __restrict__ c_idw_s, // SvdL, 20240901: not used for now..
        const TF* const __restrict__ db, const TF* const __restrict__ di, const TF* const __restrict__ z0b,
        Boundary_type bc, const TF visc, const int n_fpoints, const int n_idw_loc,
        const int icells, const int ijcells,
        const double dt)
    {
        const int n    = blockIdx.x*blockDim.x + threadIdx.x;

        const int rdim = 9;                                        
        const TF  dtf  = static_cast<TF>(dt); // SvdL, 20240901: seems like ugly solution, see how to improve later.
 
        TF u_ip_la;
        TF v_ip_la;
        TF w_ip_la;
        TF u_fp_la;
        TF v_fp_la;
        TF w_fp_la;

        if (n < n_fpoints)
        {
            const int ijkf = gi[n] + gj[n] * icells + gk[n] * ijcells; // field location of forcing point
            const TF r11 = rot[rdim * n];                              // this is maybe a redundant type defintion, since incoming rot is already type <TF>
            const TF r12 = rot[rdim * n + 1];
            const TF r13 = rot[rdim * n + 2];
            const TF r21 = rot[rdim * n + 3];
            const TF r22 = rot[rdim * n + 4];
            const TF r23 = rot[rdim * n + 5];
            const TF r31 = rot[rdim * n + 6];
            const TF r32 = rot[rdim * n + 7];
            const TF r33 = rot[rdim * n + 8];

            TF u_ip = TF(0.);
            TF v_ip = TF(0.);
            TF w_ip = TF(0.);

            // 1. interpolate surroundings neighbours to interpolation point
            for (int i = 0; i < n_idw_loc; ++i)
            {
                const int ii = i + n * n_idw_loc;
                const int ijku = ipui[ii] + ipuj[ii] * icells + ipuk[ii] * ijcells;
                const int ijkv = ipvi[ii] + ipvj[ii] * icells + ipvk[ii] * ijcells;
                const int ijkw = ipwi[ii] + ipwj[ii] * icells + ipwk[ii] * ijcells;
                
                // Do the correction based on the auxiliary velocity (i.e. intermediate velocity at next timestep without pressure forcing).
                u_ip += c_idw_u[ii] * (fld_u[ijku] + dtf * tend_u[ijku] );
                v_ip += c_idw_v[ii] * (fld_v[ijkv] + dtf * tend_v[ijkv] );
                w_ip += c_idw_w[ii] * (fld_w[ijkw] + dtf * tend_w[ijkw] );
            }

            // 2. rotate velocities to locally align with surface tangent (under the assumption that flow at second layer still aligns)
            u_ip_la = r11 * u_ip + r12 * v_ip + r13 * w_ip;
            v_ip_la = r21 * u_ip + r22 * v_ip + r23 * w_ip;
            w_ip_la = r31 * u_ip + r32 * v_ip + r33 * w_ip;

            // for now, (1) neglect flow rotation over height, (2) neglect stability effects (requires "fine enough" grid),
            // (3) assume both points are in logarithmic layer, and (4) assume zero-valued Dirichlet conditions for momentum (i.e. no-slip condition)
            // future options: investigate use of Van Driest (1956) correction and/or DNS mode.
            if (db[n] > z0b[n])
            {
                // 3. calculate (locally-aligned) velocity at forcing point
                u_fp_la = u_ip_la * std::log(db[n] / z0b[n]) / std::log(di[n] / z0b[n]);
                v_fp_la = v_ip_la * std::log(db[n] / z0b[n]) / std::log(di[n] / z0b[n]);
                w_fp_la = w_ip_la * fm::pow2(db[n] / di[n]);

                // 4. rotate back to standard grid alginment (only one component is needed here), 
                // AND overwrite old tendency at forcing point with new one to achieve this.
                tend_u[ijkf] = ( (r11 * u_fp_la + r21 * v_fp_la + r31 * w_ip_la) - fld_u[ijkf] ) / dtf;
            }
            else // SvdL, 20240901: investigate if can be changed into Van Driest like correction..
            {
                tend_u[ijkf] = ( TF(0.) - fld_u[ijkf] ) / dtf;
            }

        }
    }

    template<typename TF> __global__
    void set_forcing_points_v_g(
        const TF* const __restrict__ tend_u,
        TF* const __restrict__ tend_v,
        const TF* const __restrict__ tend_w,
        const TF* const __restrict__ fld_u,
        const TF* const __restrict__ fld_v,
        const TF* const __restrict__ fld_w,
        const TF* const __restrict__ boundary_value,
        const int* const __restrict__ gi, const int* const __restrict__ gj, const int* const __restrict__ gk,
        const TF* const __restrict__ rot,
        const int* const __restrict__ ipui, const int* const __restrict__ ipuj, const int* const __restrict__ ipuk, const TF* const __restrict__ c_idw_u,
        const int* const __restrict__ ipvi, const int* const __restrict__ ipvj, const int* const __restrict__ ipvk, const TF* const __restrict__ c_idw_v, 
        const int* const __restrict__ ipwi, const int* const __restrict__ ipwj, const int* const __restrict__ ipwk, const TF* const __restrict__ c_idw_w,
        const int* const __restrict__ ipsi, const int* const __restrict__ ipsj, const int* const __restrict__ ipsk, const TF* const __restrict__ c_idw_s, 
        const TF* const __restrict__ db, const TF* const __restrict__ di, const TF* const __restrict__ z0b,
        Boundary_type bc, const TF visc, const int n_fpoints, const int n_idw_loc,
        const int icells, const int ijcells,
        const double dt)
    {
        const int n    = blockIdx.x*blockDim.x + threadIdx.x;

        const int rdim = 9;                                       
        const TF  dtf  = static_cast<TF>(dt); // SvdL, 20240901: seems like ugly solution, see how to improve later.

        TF u_ip_la;
        TF v_ip_la;
        TF w_ip_la;
        TF u_fp_la;
        TF v_fp_la;
        TF w_fp_la;

        // Loop over all points to be forced
        if (n < n_fpoints)
        {
            const int ijkf = gi[n] + gj[n] * icells + gk[n] * ijcells; // field location of forcing point
            const TF r11 = rot[rdim * n];
            const TF r12 = rot[rdim * n + 1];
            const TF r13 = rot[rdim * n + 2];
            const TF r21 = rot[rdim * n + 3];
            const TF r22 = rot[rdim * n + 4];
            const TF r23 = rot[rdim * n + 5];
            const TF r31 = rot[rdim * n + 6];
            const TF r32 = rot[rdim * n + 7];
            const TF r33 = rot[rdim * n + 8];

            TF u_ip = TF(0.);
            TF v_ip = TF(0.);
            TF w_ip = TF(0.);

            // 1. interpolate surroundings neighbours to interpolation point
            for (int i = 0; i < n_idw_loc; ++i)
            {
                const int ii = i + n * n_idw_loc;                                  
                const int ijku = ipui[ii] + ipuj[ii] * icells + ipuk[ii] * ijcells;
                const int ijkv = ipvi[ii] + ipvj[ii] * icells + ipvk[ii] * ijcells;
                const int ijkw = ipwi[ii] + ipwj[ii] * icells + ipwk[ii] * ijcells;

                // Do the correction based on the auxiliary velocity (i.e. intermediate velocity at next timestep without pressure forcing).
                u_ip += c_idw_u[ii] * (fld_u[ijku] + dtf * tend_u[ijku] );
                v_ip += c_idw_v[ii] * (fld_v[ijkv] + dtf * tend_v[ijkv] );
                w_ip += c_idw_w[ii] * (fld_w[ijkw] + dtf * tend_w[ijkw] );
            }

            // 2. rotate velocities to locally align with surface tangent (under the assumption that flow at second layer still aligns)
            u_ip_la = r11 * u_ip + r12 * v_ip + r13 * w_ip;
            v_ip_la = r21 * u_ip + r22 * v_ip + r23 * w_ip;
            w_ip_la = r31 * u_ip + r32 * v_ip + r33 * w_ip;

            // for now, (1) neglect flow rotation over height, (2) neglect stability effects (requires "fine enough" grid),
            // (3) assume both points are in logarithmic layer, and (4) assume zero-valued Dirichlet conditions for momentum (i.e. no-slip condition)
            // future options: investigate use of Van Driest (1956) correction and/or DNS mode.
            if (db[n] > z0b[n])
            {
                // 3. calculate (locally-aligned) velocity at forcing point
                u_fp_la = u_ip_la * std::log(db[n] / z0b[n]) / std::log(di[n] / z0b[n]);
                v_fp_la = v_ip_la * std::log(db[n] / z0b[n]) / std::log(di[n] / z0b[n]);
                w_fp_la = w_ip_la *  fm::pow2(db[n] / di[n]);

                // 4. rotate back to standard grid alginment (only one component is needed here), 
                // AND overwrite old tendency at forcing point with new one to achieve this.
                tend_v[ijkf] = ( (r12 * u_fp_la + r22 * v_fp_la + r32 * w_fp_la) - fld_v[ijkf] ) / dtf;
            }
            else // SvdL, 29-06-2023: change later into Van Driest like correction..
            {
                tend_v[ijkf] = ( TF(0.) - fld_v[ijkf] ) / dtf;
            }
        }
    }

    template<typename TF> __global__
    void set_forcing_points_w_g(
        const TF* const __restrict__ tend_u,
        const TF* const __restrict__ tend_v,
        TF* const __restrict__ tend_w,
        const TF* const __restrict__ fld_u,
        const TF* const __restrict__ fld_v,
        const TF* const __restrict__ fld_w,
        const TF* const __restrict__ boundary_value,
        const int* const __restrict__ gi, const int* const __restrict__ gj, const int* const __restrict__ gk,
        const TF* const __restrict__ rot,
        const int* const __restrict__ ipui, const int* const __restrict__ ipuj, const int* const __restrict__ ipuk, const TF* const __restrict__ c_idw_u,
        const int* const __restrict__ ipvi, const int* const __restrict__ ipvj, const int* const __restrict__ ipvk, const TF* const __restrict__ c_idw_v, 
        const int* const __restrict__ ipwi, const int* const __restrict__ ipwj, const int* const __restrict__ ipwk, const TF* const __restrict__ c_idw_w,
        const int* const __restrict__ ipsi, const int* const __restrict__ ipsj, const int* const __restrict__ ipsk, const TF* const __restrict__ c_idw_s, // SvdL, 20240901: not used for now..
        const TF* const __restrict__ db, const TF* const __restrict__ di, const TF* const __restrict__ z0b,
        Boundary_type bc, const TF visc, const int n_fpoints, const int n_idw_loc,
        const int icells, const int ijcells,
        const double dt)
    {   
        const int n    = blockIdx.x*blockDim.x + threadIdx.x;

        const int rdim = 9;                                        
        const TF  dtf  = static_cast<TF>(dt); // SvdL, 20240901: seems like ugly solution, see how to improve later.

        TF u_ip_la;
        TF v_ip_la;
        TF w_ip_la;
        TF u_fp_la;
        TF v_fp_la;
        TF w_fp_la;

        // Loop over all points to be forced
        if (n < n_fpoints)
        {
            const int ijkf = gi[n] + gj[n] * icells + gk[n] * ijcells; // field location of forcing point
            const TF r11 = rot[rdim * n];
            const TF r12 = rot[rdim * n + 1];
            const TF r13 = rot[rdim * n + 2];
            const TF r21 = rot[rdim * n + 3];
            const TF r22 = rot[rdim * n + 4];
            const TF r23 = rot[rdim * n + 5];
            const TF r31 = rot[rdim * n + 6];
            const TF r32 = rot[rdim * n + 7];
            const TF r33 = rot[rdim * n + 8];

            TF u_ip = TF(0.);
            TF v_ip = TF(0.);
            TF w_ip = TF(0.);

            // 1. interpolate surroundings neighbours to interpolation point
            for (int i = 0; i < n_idw_loc; ++i)
            {
                const int ii = i + n * n_idw_loc;                                  
                const int ijku = ipui[ii] + ipuj[ii] * icells + ipuk[ii] * ijcells;
                const int ijkv = ipvi[ii] + ipvj[ii] * icells + ipvk[ii] * ijcells;
                const int ijkw = ipwi[ii] + ipwj[ii] * icells + ipwk[ii] * ijcells;

                // Do the correction based on the auxiliary velocity (i.e. intermediate velocity at next timestep without pressure forcing).
                u_ip += c_idw_u[ii] * (fld_u[ijku] + dtf * tend_u[ijku] );
                v_ip += c_idw_v[ii] * (fld_v[ijkv] + dtf * tend_v[ijkv] );
                w_ip += c_idw_w[ii] * (fld_w[ijkw] + dtf * tend_w[ijkw] );
            }

            // 2. rotate velocities to locally align with surface tangent (under the assumption that flow at second layer still aligns)
            u_ip_la = r11 * u_ip + r12 * v_ip + r13 * w_ip;
            v_ip_la = r21 * u_ip + r22 * v_ip + r23 * w_ip;
            w_ip_la = r31 * u_ip + r32 * v_ip + r33 * w_ip;

            // for now, (1) neglect flow rotation over height, (2) neglect stability effects (requires "fine enough" grid),
            // (3) assume both points are in logarithmic layer, and (4) assume zero-valued Dirichlet conditions for momentum (i.e. no-slip condition)
            // future options: investigate use of Van Driest (1956) correction and/or DNS mode.
            if (db[n] > z0b[n])
            {
                // 3. calculate (locally-aligned) velocity at forcing point
                u_fp_la = u_ip_la * std::log(db[n] / z0b[n]) / std::log(di[n] / z0b[n]);
                v_fp_la = v_ip_la * std::log(db[n] / z0b[n]) / std::log(di[n] / z0b[n]);
                w_fp_la = w_ip_la * fm::pow2(db[n] / di[n]);

                // 4. rotate back to standard grid alginment (only one component is needed here), 
                // AND overwrite old tendency at forcing point with new one to achieve this.
                tend_w[ijkf] = ( (r13 * u_fp_la + r23 * v_fp_la + r33 * w_fp_la) - fld_w[ijkf] ) / dtf;
            }
            else // SvdL, 29-06-2023: change later into Van Driest like correction..
            {
                tend_w[ijkf] = ( TF(0.) - fld_w[ijkf] ) /dtf;
            }
        }
    }

    template<typename TF> __global__
    void set_forcing_points_c_g(
        TF* const __restrict__ tend_c,
        const TF* const __restrict__ fld_c,
        const TF* const __restrict__ fld_u,
        const TF* const __restrict__ fld_v,
        const TF* const __restrict__ fld_w,
        const TF* const __restrict__ boundary_value,
        const int* const __restrict__ gi, const int* const __restrict__ gj, const int* const __restrict__ gk,
        const TF* const __restrict__ rot,
        const int* const __restrict__ ipui, const int* const __restrict__ ipuj, const int* const __restrict__ ipuk, const TF* const __restrict__ c_idw_u,
        const int* const __restrict__ ipvi, const int* const __restrict__ ipvj, const int* const __restrict__ ipvk, const TF* const __restrict__ c_idw_v, 
        const int* const __restrict__ ipwi, const int* const __restrict__ ipwj, const int* const __restrict__ ipwk, const TF* const __restrict__ c_idw_w,
        const int* const __restrict__ ipsi, const int* const __restrict__ ipsj, const int* const __restrict__ ipsk, const TF* const __restrict__ c_idw_s,
        const TF* const __restrict__ db, const TF* const __restrict__ di, const TF* const __restrict__ z0b,
        Boundary_type bc, const TF visc, const int n_fpoints, const int n_idw_loc,
        const int icells, const int ijcells, 
        const double dt)
    {
        const int n    = blockIdx.x*blockDim.x + threadIdx.x;

        const TF  dtf  = static_cast<TF>(dt); // SvdL, 20240901: seems like ugly solution, see how to improve later.

        // For Dirichlet BCs
        if (bc == Boundary_type::Dirichlet_type)
        {
            // Loop over all points to be forced
            if (n < n_fpoints)
            {
                const int ijkf = gi[n] + gj[n] * icells + gk[n] * ijcells; // field location of forcing point

                TF c_ip = TF(0.);

                // 1. interpolate surroundings neighbours to interpolation point
                for (int i = 0; i < n_idw_loc; ++i)
                {
                    const int ii = i + n * n_idw_loc;                                  
                    const int ijki = ipsi[ii] + ipsj[ii] * icells + ipsk[ii] * ijcells;
                    c_ip += c_idw_s[ii] * (fld_c[ijki] + dtf * tend_c[ijki] );
                }

                // 2. calculate scalar at forcing point and force at once
                // for now, (1) neglect stability effects (requires "fine enough" grid), and (2) assume both points are in logarithmic layer
                if (db[n] > z0b[n])
                {
                    tend_c[ijkf] = ( ( (c_ip - boundary_value[n]) * std::log(db[n] / z0b[n]) / std::log( di[n] / z0b[n]) + boundary_value[n] ) - fld_c[ijkf] ) / dtf;
                }
                else
                {
                    tend_c[ijkf] = ( boundary_value[n] - fld_c[ijkf] ) / dtf;
                }
            }
        }
        else if (bc == Boundary_type::Flux_type)
        {
            return; // SvdL, 20240918: still implement later
        }
        else if (bc == Boundary_type::Neumann_type)
        {
            return; // SvdL, 20240918: still implement later
        }
    }

    template<typename TF> __global__
    void set_forcing_points_evisc_g(
        TF* const __restrict__ fld_evisc,
        const TF* const __restrict__ fld_u,
        const TF* const __restrict__ fld_v,
        const TF* const __restrict__ fld_w,
        // const TF* const __restrict__ boundary_value, //<< SvdL, 20240909: for now not needed for eddy viscosity 
        const int* const __restrict__ gi, const int* const __restrict__ gj, const int* const __restrict__ gk,
        const TF* const __restrict__ rot,
        const int* const __restrict__ ipui, const int* const __restrict__ ipuj, const int* const __restrict__ ipuk, const TF* const __restrict__ c_idw_u,
        const int* const __restrict__ ipvi, const int* const __restrict__ ipvj, const int* const __restrict__ ipvk, const TF* const __restrict__ c_idw_v, 
        const int* const __restrict__ ipwi, const int* const __restrict__ ipwj, const int* const __restrict__ ipwk, const TF* const __restrict__ c_idw_w,
        const int* const __restrict__ ipsi, const int* const __restrict__ ipsj, const int* const __restrict__ ipsk, const TF* const __restrict__ c_idw_s,
        const TF* const __restrict__ db, const TF* const __restrict__ di, const TF* const __restrict__ z0b,
        Boundary_type bc, const TF visc, const int n_fpoints, const int n_idw_loc,
        const int icells, const int ijcells)
    {
        const int n    = blockIdx.x*blockDim.x + threadIdx.x;

        const int rdim = 9;

        TF u_ip_la;
        TF v_ip_la;
        TF w_ip_la;
        TF umag_ip_la;
        // TF ustar_ip;

        // Loop over all points to be forced
        if (n < n_fpoints)
        {
            const int ijkf = gi[n] + gj[n] * icells + gk[n] * ijcells; // field location of forcing point
            const TF r11 = rot[rdim * n];
            const TF r12 = rot[rdim * n + 1];
            const TF r13 = rot[rdim * n + 2];
            const TF r21 = rot[rdim * n + 3];
            const TF r22 = rot[rdim * n + 4];
            const TF r23 = rot[rdim * n + 5];
            const TF r31 = rot[rdim * n + 6];
            const TF r32 = rot[rdim * n + 7];
            const TF r33 = rot[rdim * n + 8];

            TF u_ip = TF(0.);
            TF v_ip = TF(0.);
            TF w_ip = TF(0.);

            // 1. interpolate surroundings neighbours to interpolation point
            for (int i = 0; i < n_idw_loc; ++i)
            {
                const int ii = i + n * n_idw_loc;                                  
                const int ijku = ipui[ii] + ipuj[ii] * icells + ipuk[ii] * ijcells;
                const int ijkv = ipvi[ii] + ipvj[ii] * icells + ipvk[ii] * ijcells;
                const int ijkw = ipwi[ii] + ipwj[ii] * icells + ipwk[ii] * ijcells;

                u_ip += c_idw_u[ii] * fld_u[ijku];
                v_ip += c_idw_v[ii] * fld_v[ijkv];
                w_ip += c_idw_w[ii] * fld_w[ijkw];
            }

            // 2. rotate velocities to locally align with surface tangent (under the assumption that flow at second layer still aligns)
            u_ip_la = r11 * u_ip + r12 * v_ip + r13 * w_ip;
            v_ip_la = r21 * u_ip + r22 * v_ip + r23 * w_ip;

            umag_ip_la = std::pow(fm::pow2(u_ip_la) + fm::pow2(v_ip_la), TF(0.5));

            if (db[n] > z0b[n])
            {
                // 3. calculate local shear velocity (ustar) and thereby eddy viscosity at forcing point
                // for now, (1) neglect flow rotation over height, (2) neglect stability effects (requires "fine enough" grid),
                // (3) assume both points are in logarithmic layer, and (4) assume zero-valued Dirichlet conditions for momentum (i.e. no-slip condition)
                // future options: investigate use of Van Driest (1956) correction and/or DNS mode.
                fld_evisc[ijkf] = fm::pow2(Constants::kappa<TF>) * umag_ip_la * db[n] / std::log(di[n] / z0b[n]);
            }
            else
            {
                // SvdL, 20240901: Currently defaulting to WRONG(?) evisc-value due to zero boundary distance..
                fld_evisc[ijkf] = fm::pow2(Constants::kappa<TF>) * umag_ip_la * z0b[n] / std::log(di[n] / z0b[n]);
            }
        }
    }


    template<typename TF> __global__
    void set_ib_points_g(
        TF* const __restrict__ tend_var,
        const TF* const __restrict__ var,
        const int* const __restrict__ ijk_ib, 
        const TF val,
        const int n_ib,
        const double dt)
    {
        const int n    = blockIdx.x*blockDim.x + threadIdx.x;
        const TF  dtf  = static_cast<TF>(dt); // SvdL, 20240901: seems like ugly solution, see how to improve later.

        if (n < n_ib)
        {
            tend_var[ijk_ib[n]] = ( val - var[ijk_ib[n]] ) / dtf;       
        }

    }

    template<typename TF> __global__
    void set_ib_points_evisc_g(
        TF* const __restrict__ var,
        const int* const __restrict__ ijk_ib, 
        const TF val,
        const int n_ib)
    {
        const int n = blockIdx.x*blockDim.x + threadIdx.x;

        if (n < n_ib)
            var[ijk_ib[n]] = val;
        
    }
}

// SvdL, 20240908: started CUDA implementation
#ifdef USECUDA

template <typename TF>
void Immersed_boundary<TF>::exec_viscosity()
{
    if (sw_ib == IB_type::Disabled)
    return;

    const int blocki = 256;

    const int n_fp_c = fpoints.at("s").n_fpoints;
    const int n_ib_c = fpoints.at("s").n_ibpoints;

    const int gridfp_c = n_fp_c / blocki + (n_fp_c % blocki > 0);
    const int gridib_c = n_ib_c / blocki + (n_ib_c % blocki > 0);

    dim3 gridGPU_fp_c(gridfp_c);
    dim3 gridGPU_ib_c(gridib_c);

    dim3 blockGPU(blocki);

    set_forcing_points_evisc_g<TF><<gridGPU_fp_c, blockGPU>>>(
            fields.sd.at("evisc")->fld_g,
            fields.mp.at("u")->fld_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("w")->fld_g,
            // fpoints.at("s").sbot.at("s")_g,                                                                       //<< SvdL, 20240909: for now not needed here,  // value of boundary conditions to be enforced
            fpoints.at("s").i_g, fpoints.at("s").j_g, fpoints.at("s").k_g,                                           // points to be forced
            fpoints.at("s").rot_g,                                                                                   // rotational matrix for local surface alignment
            fpoints.at("s").ip_u_i_g, fpoints.at("s").ip_u_j_g, fpoints.at("s").ip_u_k_g, fpoints.at("s").c_idw_u_g, // locations of the neighbouring u-points + weights
            fpoints.at("s").ip_v_i_g, fpoints.at("s").ip_v_j_g, fpoints.at("s").ip_v_k_g, fpoints.at("s").c_idw_v_g, // locations of the neighbouring v-points + weights
            fpoints.at("s").ip_w_i_g, fpoints.at("s").ip_w_j_g, fpoints.at("s").ip_w_k_g, fpoints.at("s").c_idw_w_g, // locations of the neighbouring w-points + weights
            fpoints.at("s").ip_s_i_g, fpoints.at("s").ip_s_j_g, fpoints.at("s").ip_s_k_g, fpoints.at("s").c_idw_s_g, // locations of the neighbouring s-points + weights
            fpoints.at("s").dist_b_g,                                                                                // distance nearest immersed boundary point to forcing point
            fpoints.at("s").dist_i_g,                                                                                // distance interpolation point to forcing point
            fpoints.at("s").z0b_g,                                                                                   // local roughness lengths of forcing points (all scalars will have same for now..)
            Boundary_type::Dirichlet_type,                                                                           // should contain Boundary_Type:: for all scalars (make variation between scalars possible?), also unused for evisc
            fields.visc, n_fp_c, this->n_idw_points,
            gd.icells, gd.ijcells);
    cuda_check_error();

    // eddy viscosity should just be zero inside objects
    set_ib_points_evisc_g<TF><<gridGPU_fib_c, blockGPU>>>(
            fields.sd.at("evisc")->fld_g,
            ibpoints.at("s").ijk._g, TF(0.), 
            n_ib_c);
    cuda_check_error();

    // Enforce cyclic boundary conditions for updated evisc
    boundary_cyclic.exec_g(fields.sd.at("evisc")->fld_g);

}

template <typename TF>
void Immersed_boundary<TF>::exec(const double dt)
{
    if (sw_ib == IB_type::Disabled)
    return;

    const int blocki = 256;

    const int n_fp_u = fpoints.at("u").n_fpoints;
    const int n_ib_u = fpoints.at("u").n_ibpoints;
    const int n_fp_v = fpoints.at("v").n_fpoints;
    const int n_ib_v = fpoints.at("v").n_ibpoints;
    const int n_fp_w = fpoints.at("w").n_fpoints;
    const int n_ib_w = fpoints.at("w").n_ibpoints;
    const int n_fp_c = fpoints.at("s").n_fpoints;
    const int n_ib_c = fpoints.at("s").n_ibpoints;

    const int gridfp_u = n_fp_u / blocki + (n_fp_u % blocki > 0);
    const int gridib_u = n_ib_u / blocki + (n_ib_u % blocki > 0);
    const int gridfp_v = n_fp_v / blocki + (n_fp_v % blocki > 0);
    const int gridib_v = n_ib_v / blocki + (n_ib_v % blocki > 0);
    const int gridfp_w = n_fp_w / blocki + (n_fp_w % blocki > 0);
    const int gridib_w = n_ib_w / blocki + (n_ib_w % blocki > 0);
    const int gridfp_c = n_fp_c / blocki + (n_fp_c % blocki > 0);
    const int gridib_c = n_ib_c / blocki + (n_ib_c % blocki > 0);

    dim3 gridGPU_fp_u(gridfp_u);
    dim3 gridGPU_ib_u(gridib_u);
    dim3 gridGPU_fp_v(gridfp_v);
    dim3 gridGPU_ib_v(gridib_v);
    dim3 gridGPU_fp_w(gridfp_w);       
    dim3 gridGPU_ib_w(gridib_w);
    dim3 gridGPU_fp_c(gridfp_c);
    dim3 gridGPU_ib_c(gridib_c);

    dim3 blockGPU(blocki);

    set_forcing_points_u_g<TF><<<gridGPU_fp_u, blockGPU>>>(
            fields.mt.at("u")->fld_g,
            fields.mt.at("v")->fld_g,
            fields.mt.at("w")->fld_g,
            fields.mp.at("u")->fld_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("w")->fld_g,
            fpoints.at("u").mbot_g,                                                                                            // value of boundary conditions to be enforced
            fpoints.at("u").i_g, fpoints.at("u").j_g, fpoints.at("u").k_g,                                           // points to be forced
            fpoints.at("u").rot_g,                                                                                             // rotational matrix for local surface alignment
            fpoints.at("u").ip_u_i_g, fpoints.at("u").ip_u_j_g, fpoints.at("u").ip_u_k_g, fpoints.at("u").c_idw_u_g, // locations of the neighbouring u-points + weights
            fpoints.at("u").ip_v_i_g, fpoints.at("u").ip_v_j_g, fpoints.at("u").ip_v_k_g, fpoints.at("u").c_idw_v_g, // locations of the neighbouring v-points + weights
            fpoints.at("u").ip_w_i_g, fpoints.at("u").ip_w_j_g, fpoints.at("u").ip_w_k_g, fpoints.at("u").c_idw_w_g, // locations of the neighbouring w-points + weights
            fpoints.at("u").ip_s_i_g, fpoints.at("u").ip_s_j_g, fpoints.at("u").ip_s_k_g, fpoints.at("u").c_idw_s_g, // locations of the neighbouring s-points + weights
            fpoints.at("u").dist_b_g,                                                                                              // distance nearest immersed boundary point to forcing point
            fpoints.at("u").dist_i_g,                                                                                              // distance interpolation point to forcing point
            fpoints.at("u").z0b_g,                                                                                             // local roughness lengths of forcing points
            Boundary_type::Dirichlet_type,                                                                                          // only allow no-slip conditions for momentum (for now..)
            fields.visc, n_fp_u, this->n_idw_points,
            gd.icells, gd.ijcells,
            dt);
    cuda_check_error();

    set_ib_points_g<TF><<<gridGPU_ib_u, blockGPU>>>(
            fields.mt.at("u")->fld_g,
            fields.mp.at("u")->fld_g,
            ibpoints.at("u").ijk_g, TF(0.), 
            n_ib_u,
            dt);
    cuda_check_error();

    set_forcing_points_v_g<TF><<<gridGPU_fp_v, blockGPU>>>(
            fields.mt.at("u")->fld_g,
            fields.mt.at("v")->fld_g,
            fields.mt.at("w")->fld_g,
            fields.mp.at("u")->fld_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("w")->fld_g,
            fpoints.at("v").mbot_g,                                                                    // value of boundary conditions to be enforced
            fpoints.at("v").i_g, fpoints.at("v").j_g, fpoints.at("v").k_g,                   // points to be forced
            fpoints.at("v").rot_g,                                                                     // rotational matrix for local surface alignment
            fpoints.at("v").ip_u_i_g, fpoints.at("u").ip_u_j_g, fpoints.at("v").ip_u_k_g, fpoints.at("v").c_idw_u_g, // locations of the neighbouring u-points + weights
            fpoints.at("v").ip_v_i_g, fpoints.at("v").ip_v_j_g, fpoints.at("v").ip_v_k_g, fpoints.at("v").c_idw_v_g, // locations of the neighbouring v-points + weights
            fpoints.at("v").ip_w_i_g, fpoints.at("v").ip_w_j_g, fpoints.at("v").ip_w_k_g, fpoints.at("v").c_idw_w_g, // locations of the neighbouring w-points + weights
            fpoints.at("v").ip_s_i_g, fpoints.at("v").ip_s_j_g, fpoints.at("v").ip_s_k_g, fpoints.at("v").c_idw_s_g, // locations of the neighbouring s-points + weights
            fpoints.at("v").dist_b_g,                                                                      // distance nearest immersed boundary point to forcing point
            fpoints.at("v").dist_i_g,                                                                      // distance interpolation point to forcing point
            fpoints.at("v").z0b_g,                                                                     // local roughness lengths of forcing points
            Boundary_type::Dirichlet_type,                                                                  // only allow no-slip conditions for momentum (for now..)
            fields.visc, n_fp_v, this->n_idw_points,
            gd.icells, gd.ijcells, 
            dt);
    cuda_check_error();

    set_ib_points_g<TF><<<gridGPU_ib_v, blockGPU>>>(
            fields.mt.at("v")->fld_g,
            fields.mp.at("v")->fld_g,
            ibpoints.at("v").ijk_g, TF(0.), 
            n_ib_v,
            dt);
    cuda_check_error();

    set_forcing_points_w_g<TF><<<gridGPU_fp_w, blockGPU>>>(
            fields.mt.at("u")->fld_g,
            fields.mt.at("v")->fld_g,
            fields.mt.at("w")->fld_g,
            fields.mp.at("u")->fld_g,
            fields.mp.at("v")->fld_g,
            fields.mp.at("w")->fld_g,
            fpoints.at("w").mbot_g,                                                                    // value of boundary conditions to be enforced
            fpoints.at("w").i_g, fpoints.at("w").j_g, fpoints.at("w").k_g,                   // points to be forced
            fpoints.at("w").rot_g,                                                                     // rotational matrix for local surface alignment
            fpoints.at("w").ip_u_i_g, fpoints.at("w").ip_u_j_g, fpoints.at("w").ip_u_k_g, fpoints.at("u").c_idw_u_g, // locations of the neighbouring u-points + weights
            fpoints.at("w").ip_v_i_g, fpoints.at("w").ip_v_j_g, fpoints.at("w").ip_v_k_g, fpoints.at("u").c_idw_v_g, // locations of the neighbouring v-points + weights
            fpoints.at("w").ip_w_i_g, fpoints.at("w").ip_w_j_g, fpoints.at("w").ip_w_k_g, fpoints.at("u").c_idw_w_g, // locations of the neighbouring w-points + weights
            fpoints.at("w").ip_s_i_g, fpoints.at("w").ip_s_j_g, fpoints.at("w").ip_s_k_g, fpoints.at("u").c_idw_s_g, // locations of the neighbouring s-points + weights
            fpoints.at("w").dist_b_g,                                                                      // distance nearest immersed boundary point to forcing point
            fpoints.at("w").dist_i_g,                                                                      // distance interpolation point to forcing point
            fpoints.at("w").z0b_g,                                                                     // local roughness lengths of forcing points
            Boundary_type::Dirichlet_type,                                                                  // only allow no-slip conditions for momentum (for now..)
            fields.visc, n_fp_w, this->n_idw_points,
            gd.icells, gd.ijcells,
            dt);
    cuda_check_error();    

    set_ib_points_g<TF><<<gridGPU_ib_w, blockGPU>>>(
            fields.mt.at("w")->fld_g,
            fields.mp.at("w")->fld_g,
            ibpoints.at("w").ijk_g, TF(0.), 
            n_ib_w,
            dt);
    cuda_check_error();

    for (auto &it : fields.sp)
    {
        set_forcing_points_c_g<TF><<<gridGPU_fp_c, blockGPU>>>(
                fields.st.at(it.first)->fld_g,
                it.second->fld_g,
                fields.mp.at("u")->fld_g,
                fields.mp.at("v")->fld_g,
                fields.mp.at("w")->fld_g,
                fpoints.at("s").sbot.at(it.first)_g,                                                       // value of boundary conditions to be enforced
                fpoints.at("s").i_g, fpoints.at("s").j_g, fpoints.at("s").k_g,                   // points to be forced
                fpoints.at("s").rot_g,                                                                     // rotational matrix for local surface alignment (although not used yet for scalars: DO calculate, needed for evisc)
                fpoints.at("s").ip_u_i_g, fpoints.at("s").ip_u_j_g, fpoints.at("s").ip_u_k_g, fpoints.at("s").c_idw_u_g, // locations of the neighbouring u-points + weights
                fpoints.at("s").ip_v_i_g, fpoints.at("s").ip_v_j_g, fpoints.at("s").ip_v_k_g, fpoints.at("s").c_idw_v_g, // locations of the neighbouring v-points + weights
                fpoints.at("s").ip_w_i_g, fpoints.at("s").ip_w_j_g, fpoints.at("s").ip_w_k_g, fpoints.at("s").c_idw_w_g, // locations of the neighbouring w-points + weights
                fpoints.at("s").ip_s_i_g, fpoints.at("s").ip_s_j_g, fpoints.at("s").ip_s_k_g, fpoints.at("s").c_idw_s_g, // locations of the neighbouring s-points + weights
                fpoints.at("s").dist_b_g,                                                                      // distance nearest immersed boundary point to forcing point
                fpoints.at("s").dist_i_g,                                                                      // distance interpolation point to forcing point
                fpoints.at("s").z0b_g,                                                                     // local roughness lengths of forcing points (all scalars will have same for now..)
                sbc.at(it.first),                                                                                         // should contain Boundary_Type:: for all scalars (make variation between scalars possible?)
                fields.visc, n_fp_c, this->n_idw_points,
                gd.icells, gd.ijcells,
                dt);
        cuda_check_error();

        set_ib_points_g<TF><<<gridGPU_ib_c, blockGPU>>>(
                fields.st.at(it.first)->fld_g,
                it.second->fld_g,
                ibpoints.at("s").ijk_g, TF(0.), 
                n_ib_c,
                dt); 
        cuda_check_error(); 
    }  
}
#endif

template <typename TF>
void Immersed_boundary<TF>::prepare_device()
{
    if (sw_ib == IB_type::Disabled)
        return;

    // Allocate and forward FP information (forcing points)
    for (auto& fp : fpoints)
    {
        const int n_fpoints = fp.second.n_fpoints;
        const int rdim = 9;

        const int imemsize_1d = n_fpoints*sizeof(int);
        const int fmemsize_1d = n_fpoints*sizeof(TF);

        const int fmemsize_ro = n_fpoints*rdim*sizeof(TF);

        const int imemsize_2d = n_fpoints*n_idw_points*sizeof(int);
        const int fmemsize_2d = n_fpoints*n_idw_points*sizeof(TF);

         // Allocate
        cuda_safe_call(hipMalloc(&fp.second.i_g  , imemsize_1d));
        cuda_safe_call(hipMalloc(&fp.second.j_g  , imemsize_1d));
        cuda_safe_call(hipMalloc(&fp.second.k_g  , imemsize_1d));
        // cuda_safe_call(hipMalloc(&fp.second.ijk_g, imemsize_1d));

        cuda_safe_call(hipMalloc(&fp.second.rot_g   , fmemsize_ro));

        cuda_safe_call(hipMalloc(&fp.second.dist_b_g, fmemsize_1d));
        cuda_safe_call(hipMalloc(&fp.second.dist_i_g, fmemsize_1d));
        cuda_safe_call(hipMalloc(&fp.second.z0b_g   , fmemsize_1d));

        cuda_safe_call(hipMalloc(&fp.second.ip_u_i_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_u_j_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_u_k_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_v_i_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_v_j_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_v_k_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_w_i_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_w_j_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_w_k_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_s_i_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_s_j_g, imemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.ip_s_k_g, imemsize_2d));

        cuda_safe_call(hipMalloc(&fp.second.c_idw_u_g, fmemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.c_idw_v_g, fmemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.c_idw_w_g, fmemsize_2d));
        cuda_safe_call(hipMalloc(&fp.second.c_idw_s_g, fmemsize_2d));

        if (fp.first == "u" || fp.first == "v" || fp.first == "w")
            cuda_safe_call(hipMalloc(&fp.second.mbot_g, fmemsize_1d));
        else
            for (auto& it : fp.second.sbot)
                cuda_safe_call(hipMalloc(&fp.second.sbot_g[it.first], fmemsize_1d));

        // Forward copy
        cuda_safe_call(hipMemcpy(fp.second.i_g, fp.second.i.data(), imemsize_1d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.j_g, fp.second.j.data(), imemsize_1d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.k_g, fp.second.k.data(), imemsize_1d, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpy(fp.second.dist_b_g, fp.second.dist_b.data(), fmemsize_1d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.dist_i_g, fp.second.dist_i.data(), fmemsize_1d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.z0b_g   , fp.second.z0b.data()   , fmemsize_1d, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpy(fp.second.rot_g   , fp.second.rot.data()   , fmemsize_ro, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpy(fp.second.ip_u_i_g, fp.second.ip_u_i.data(), imemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.ip_u_j_g, fp.second.ip_u_j.data(), imemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.ip_u_k_g, fp.second.ip_u_k.data(), imemsize_2d, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpy(fp.second.ip_v_i_g, fp.second.ip_v_i.data(), imemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.ip_v_j_g, fp.second.ip_v_j.data(), imemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.ip_v_k_g, fp.second.ip_v_k.data(), imemsize_2d, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpy(fp.second.ip_w_i_g, fp.second.ip_w_i.data(), imemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.ip_w_j_g, fp.second.ip_w_j.data(), imemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.ip_w_k_g, fp.second.ip_w_k.data(), imemsize_2d, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpy(fp.second.ip_s_i_g, fp.second.ip_s_i.data(), imemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.ip_s_j_g, fp.second.ip_s_j.data(), imemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.ip_s_k_g, fp.second.ip_s_k.data(), imemsize_2d, hipMemcpyHostToDevice));

        cuda_safe_call(hipMemcpy(fp.second.c_idw_u_g, fp.second.c_idw_u.data(), fmemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.c_idw_v_g, fp.second.c_idw_v.data(), fmemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.c_idw_w_g, fp.second.c_idw_w.data(), fmemsize_2d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(fp.second.c_idw_s_g, fp.second.c_idw_s.data(), fmemsize_2d, hipMemcpyHostToDevice));

        if (fp.first == "u" || fp.first == "v" || fp.first == "w")
            cuda_safe_call(hipMemcpy(fp.second.mbot_g, fp.second.mbot.data(), fmemsize_1d, hipMemcpyHostToDevice));
        else
            for (auto& it : fp.second.sbot)
                cuda_safe_call(hipMemcpy(fp.second.sbot_g.at(it.first), fp.second.sbot.at(it.first).data(), 
                               fmemsize_1d, hipMemcpyHostToDevice));
    }

    // Allocate and forward IB information (internal points)
    for (auto& ib : ibpoints)
    {
        const int n_ibpoints = ib.second.n_ibpoints;

        const int imemsize_1d = n_ibpoints*sizeof(int);

         // Allocate
        cuda_safe_call(hipMalloc(&ib.second.i_g  , imemsize_1d));
        cuda_safe_call(hipMalloc(&ib.second.j_g  , imemsize_1d));
        cuda_safe_call(hipMalloc(&ib.second.k_g  , imemsize_1d));
        cuda_safe_call(hipMalloc(&ib.second.ijk_g, imemsize_1d));

        // Forward copy
        cuda_safe_call(hipMemcpy(ib.second.i_g  , ib.second.i.data()  , imemsize_1d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(ib.second.j_g  , ib.second.j.data()  , imemsize_1d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(ib.second.k_g  , ib.second.k.data()  , imemsize_1d, hipMemcpyHostToDevice));
        cuda_safe_call(hipMemcpy(ib.second.ijk_g, ib.second.ijk.data(), imemsize_1d, hipMemcpyHostToDevice));
    }      
}

template <typename TF>
void Immersed_boundary<TF>::clear_device()
{
    if (sw_ib == IB_type::Disabled)
        return;

    // De-allocate all FP information (forcing points)
    for (auto& fp : fpoints)
    {
        cuda_safe_call(hipFree(fp.second.i_g));
        cuda_safe_call(hipFree(fp.second.j_g));
        cuda_safe_call(hipFree(fp.second.k_g));

        cuda_safe_call(hipFree(fp.second.rot_g));

        cuda_safe_call(hipFree(fp.second.dist_b_g));
        cuda_safe_call(hipFree(fp.second.dist_i_g));
        cuda_safe_call(hipFree(fp.second.z0b_g   ));

        cuda_safe_call(hipFree(fp.second.ip_u_i_g));
        cuda_safe_call(hipFree(fp.second.ip_u_j_g));
        cuda_safe_call(hipFree(fp.second.ip_u_k_g));
        cuda_safe_call(hipFree(fp.second.ip_v_i_g));
        cuda_safe_call(hipFree(fp.second.ip_v_j_g));
        cuda_safe_call(hipFree(fp.second.ip_v_k_g));
        cuda_safe_call(hipFree(fp.second.ip_w_i_g));
        cuda_safe_call(hipFree(fp.second.ip_w_j_g));
        cuda_safe_call(hipFree(fp.second.ip_w_k_g));
        cuda_safe_call(hipFree(fp.second.ip_s_i_g));
        cuda_safe_call(hipFree(fp.second.ip_s_j_g));
        cuda_safe_call(hipFree(fp.second.ip_s_k_g));

        cuda_safe_call(hipFree(fp.second.c_idw_u_g));
        cuda_safe_call(hipFree(fp.second.c_idw_v_g));
        cuda_safe_call(hipFree(fp.second.c_idw_w_g));
        cuda_safe_call(hipFree(fp.second.c_idw_s_g));

        if (fp.first == "u" || fp.first == "v" || fp.first == "w")
            cuda_safe_call(hipFree(fp.second.mbot_g));
        else
            for (auto& it : fp.second.sbot)
                cuda_safe_call(hipFree(fp.second.sbot_g[it.first]));
    }

    // De-allocate all IB information (internal points)
    for (auto& ib : ibpoints)
    {
        cuda_safe_call(hipFree(ib.second.i_g  ));
        cuda_safe_call(hipFree(ib.second.j_g  ));
        cuda_safe_call(hipFree(ib.second.k_g  ));
        cuda_safe_call(hipFree(ib.second.ijk_g));
    }     

}

template class Immersed_boundary<double>;
template class Immersed_boundary<float>;
